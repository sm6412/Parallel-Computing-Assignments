#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void getmaxcu(unsigned int *, unsigned int *);
unsigned int getmax(unsigned int *, unsigned int);


__global__ 
void getmaxcu(unsigned int * gpu_numbers, unsigned int * gpu_max){
  extern __shared__ unsigned int block_data[];
  unsigned int tid = threadIdx.x;
  block_data[tid] = gpu_numbers[(blockIdx.x*blockDim.x) + threadIdx.x];
  __syncthreads();

  // reduce 
  for(unsigned int offset=1; offset < blockDim.x; offset *= 2) {
    int compare_val = 2*offset; 
    if (tid % (compare_val) == 0) {
      if(block_data[tid] < block_data[tid + offset]){
        block_data[tid] = block_data[tid + offset];
      }
    
    }
    __syncthreads();
  }
  // write block max to list of maxes 
  if (tid == 0){
    gpu_max[blockIdx.x] = block_data[0];
  } 
  __syncthreads();
}


int main(int argc, char *argv[])
{
  unsigned int size = 0;  // The size of the array
  unsigned int i;  // loop index
  unsigned int * numbers; //pointer to the array
	unsigned int * gpu_numbers; 
  unsigned int * host_max;
	unsigned int * gpu_max;
  unsigned int * gpu_max2;


    
  if(argc !=2)
  {
      printf("usage: maxseq num\n");
      printf("num = size of the array\n");
      exit(1);
  }
   
  size = atol(argv[1]);

  numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
  if( !numbers )
  {
      printf("Unable to allocate mem for an array of size %u\n", size);
      exit(1);
  }    

  srand(time(NULL)); // setting a seed for the random number generator
  // Fill-up the array with random numbers from 0 to size-1 
  for( i = 0; i < size; i++)
      numbers[i] = rand()  % size;

  // print sequential answer 
  //printf("Correct answer is: %u\n", getmax(numbers, size));



	// allocate gpu memory for array of randomly generated numbers 
  int allocation_size = size * sizeof(unsigned int);
  hipMalloc((void**)&gpu_numbers,allocation_size);
  hipMemcpy(gpu_numbers,numbers,allocation_size,hipMemcpyHostToDevice);
   
   // get block num, and threads per block
  int threads_per_block = 1024;
  int num_of_blocks = (int)ceil(size/(double)threads_per_block);

  // allocate gpu memory for max number 
  hipMalloc((void**)&gpu_max,num_of_blocks*sizeof(unsigned int));
  
  // find maxes of each block
  getmaxcu<<<num_of_blocks,threads_per_block,threads_per_block*sizeof(unsigned int)>>>(gpu_numbers,gpu_max);
  
  // while there is still more than 1 block, continue
  // to reduce maxes
  while(num_of_blocks>1){
    // get new number of blocks
    num_of_blocks = (int)ceil(num_of_blocks/(double)threads_per_block);
    hipMalloc((void**)&gpu_max2,num_of_blocks*sizeof(unsigned int));

    // rerun kernel
    getmaxcu<<<num_of_blocks,threads_per_block,threads_per_block*sizeof(unsigned int)>>>(gpu_max,gpu_max2); 

    // move over data
    hipMalloc((void**)&gpu_max,num_of_blocks*sizeof(unsigned int));
    hipMemcpy(gpu_max,gpu_max2,num_of_blocks*sizeof(unsigned int),hipMemcpyDeviceToDevice);
  }

  // allocate memory for max on host
  host_max = (unsigned int *)malloc(num_of_blocks * sizeof(unsigned int));
  // copy max from device to host
  hipMemcpy(host_max,gpu_max,num_of_blocks * sizeof(unsigned int),hipMemcpyDeviceToHost);

  // display max
  printf(" The maximum number in the array is: %u\n", host_max[0]);

  // free memory
  hipFree(gpu_numbers);
  hipFree(gpu_max);
  free(numbers);
  free(host_max);
  exit(0);
}

unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
  if(num[i] > max)
     max = num[i];

  return( max );

}




